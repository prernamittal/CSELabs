#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void addMatrix (double *matA,double *matB,double *matC,int length){
	int i=blockIdx.x * blockDim.x + threadIdx.x;
	int j=blockIdx.y * blockDim.y + threadIdx.y;
	int k = i+j*length;
	if(i<length&&j<length)
		matC[k] = matA[k]+matB[k];	
}

int main(void){
	
	return 0;
}